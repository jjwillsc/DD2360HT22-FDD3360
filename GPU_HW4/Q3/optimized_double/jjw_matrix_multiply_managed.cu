/*
* CUDA Problem-double (Multiple Matrix - Managed Memory)
*/

// Include Header Files
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <typeinfo>
#include <hip/hip_runtime.h>

using std::generate;

typedef double DataType;

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
  //@@ Insert code to implement matrix multiplication here

  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Boundary check
  if (!(row < numARows && col < numBColumns))
    return;

  // Iterate over row, and down column
  DataType tmp = 0;
  for (int i = 0; i < numAColumns; ++i) {
    // Accumulate results for a single element
    tmp += A[row * numAColumns + i] * B[i * numBColumns + col];
  }
  C[row * numBColumns + col] = tmp;
}

// Check result on the CPU
void gemm__(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns) {
  // For every row...
  for (int i = 0; i < numARows; ++i) {
    // For every column...
    for (int j = 0; j < numBColumns; ++j) {
      // For every element in the row-column pair
      DataType tmp = 0;
      for (int k = 0; k < numAColumns; ++k) {
        // Accumulate the partial results
        tmp += A[i * numAColumns + k] * B[k * numBColumns + j];
      }
      C[i * numBColumns + j] = tmp;
    }
  }
}

//@@ Insert code to implement timer start
clock_t st, en;
void timerStart() {
  st = clock();
}

//@@ Insert code to implement timer stop
void timerStop(char stepName[]) {
  en = clock();
  clock_t elapsed = en - st;
  printf("%s: %u ms elapsed.\n", stepName, elapsed);
}

int main(int argc, char **argv) {  
  DataType *A; // The A matrix
  DataType *B; // The B matrix
  DataType *C; // The output C matrix
  DataType *resultRef; // The reference result
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBColumns = atoi(argv[3]);
  numBRows = numAColumns;
  numCRows = numARows;
  numCColumns = numBColumns;

  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  printf("Data Type: %s\n", typeid(DataType).name());

  //@@ Insert code below to allocate memory for input and output
  size_t bytesA = numARows * numAColumns * sizeof(DataType);
  size_t bytesB = numBRows * numBColumns * sizeof(DataType);
  size_t bytesC = numCRows * numCColumns * sizeof(DataType);
  hipMallocManaged(&A, bytesA);
  hipMallocManaged(&B, bytesB);
  hipMallocManaged(&C, bytesC);

  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU
  generate(A, A + numARows * numAColumns, []() { return rand() / (DataType)RAND_MAX; });
  generate(B, B + numBRows * numBColumns, []() { return rand() / (DataType)RAND_MAX; });

  //@@ Initialize the grid and block dimensions here
  int THREADS = 32;
  int BLOCKS_ROW = (numARows + THREADS - 1) / THREADS;
  int BLOCKS_COL = (numBColumns + THREADS - 1) / THREADS;
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS_COL, BLOCKS_ROW);

  //@@ Launch the GPU Kernel here
  timerStart();
  gemm<<<blocks, threads>>>(A, B, C, numARows, numAColumns, numBRows, numBColumns);
  hipDeviceSynchronize();
  timerStop("Kernel");

  //@@ Insert code below to compare the output with the reference
  hipMallocManaged(&resultRef, bytesC);
  gemm__(A, B, resultRef, numARows, numAColumns, numBRows, numBColumns);

  for (int i = 0; i < numCRows * numCColumns; ++i)
    if (fabs(C[i] - resultRef[i]) > 1e-4) {
      printf("Wrong\n");
      break;
    }

  //@@ Free the memory here
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(resultRef);

  return 0;
}


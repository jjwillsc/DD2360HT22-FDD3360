#include "hip/hip_runtime.h"
/*
* CUDA Problem (Vector Addition)
* 
*/

// Include Header Files

#include <stdio.h>
#include <string.h>
#include <time.h>
#include <algorithm>
#include <hip/hip_runtime.h>

using std::generate;

typedef double DataType;

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
//@@ Insert code to implement vector addition here
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;

  // Boundary check
  if (idx < len)
    out[idx] = in1[idx] + in2[idx];
}

//@@ Insert code to implement timer start
clock_t st, en;
void timerStart() {
  st = clock();
}

//@@ Insert code to implement timer stop
void timerStop(char stepName[]) {
  en = clock();
  clock_t elapsed = en - st;
  printf("%s: %u ms elapsed.\n", stepName, elapsed);
}

int main(int argc, char **argv) {
  int inputLength;
  int S_seg;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  // Create CUDA Streams
  hipStream_t stream[4];
  for (int i = 0; i < 4; ++i)
    hipStreamCreate(stream + i);

  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);
  S_seg = atoi(argv[2]);

  printf("The input length is %d, and the segment size is %d\n", inputLength, S_seg);
  
  //@@ Insert code below to allocate Host memory for input and output
  size_t bytes = inputLength * sizeof(DataType);
  hipHostMalloc(&hostInput1, bytes);
  hipHostMalloc(&hostInput2, bytes);
  hipHostMalloc(&hostOutput, bytes);

  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  generate(hostInput1, hostInput1 + inputLength, []() { return (DataType)rand() / RAND_MAX; });
  generate(hostInput2, hostInput2 + inputLength, []() { return (DataType)rand() / RAND_MAX; });

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, bytes);
  hipMalloc(&deviceInput2, bytes);
  hipMalloc(&deviceOutput, bytes);

  //@@ Insert code to below to Copy memory to the GPU here
  timerStart();
  for (int p_start = 0, k = 0; p_start < inputLength; p_start += S_seg, (++k) %= 4) {
    int p_end = p_start + S_seg;
    if (p_end > inputLength)
      p_end = inputLength;
    
    hipMemcpyAsync(deviceInput1 + p_start, hostInput1 + p_start, (p_end - p_start) * sizeof(DataType), hipMemcpyHostToDevice, stream[k]);
  }
  for (int p_start = 0, k = 0; p_start < d; p_start += S_seg, (++k) %= 4) {
    int p_end = p_start + S_seg;
    if (p_end > inputLength)
      p_end = inputLength;
    
    hipMemcpyAsync(deviceInput2 + p_start, hostInput2 + p_start, (p_end - p_start) * sizeof(DataType), hipMemcpyHostToDevice, stream[k]);
  }
  timerStop("Host to Device");

  //@@ Launch the GPU Kernel here
  timerStart();
  for (int p_start = 0, k = 0; p_start < inputLength; p_start += S_seg, (++k) %= 4) {
    int p_end = p_start + S_seg;
    if (p_end > inputLength)
      p_end = inputLength;
    
    //@@ Initialize the 1D grid and block dimensions here
    int BLOCK_SIZE = 256;
    int GRID_SIZE = (p_end - p_start + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    vecAdd<<<GRID_SIZE, BLOCK_SIZE, 0, stream[k]>>>(deviceInput1 + p_start, deviceInput2 + p_start, deviceOutput + p_start, p_end - p_start);
  }
  timerStop("Kernel");

  //@@ Copy the GPU memory back to the CPU here
  timerStart();
  for (int p_start = 0, k = 0; p_start < inputLength; p_start += S_seg, (++k) %= 4) {
    int p_end = p_start + S_seg;
    if (p_end > inputLength)
      p_end = inputLength;
    
    hipMemcpyAsync(hostOutput + p_start, deviceOutput + p_start, (p_end - p_start) * sizeof(DataType), hipMemcpyDeviceToHost, stream[k]);
  }
  hipDeviceSynchronize();
  timerStop("Device To Host");

  //@@ Insert code below to compare the output with the reference
  hipHostMalloc(&resultRef, bytes);
  for (int i = 0; i < inputLength; ++i)
    resultRef[i] = hostInput1[i] + hostInput2[i];

  for (int i = 0; i < inputLength; ++i)
    if (fabs(hostOutput[i] - resultRef[i]) > 1e-6) {
      printf("Wrong\n");
      break;
    }

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  hipFree(hostInput1);
  hipFree(hostInput2);
  hipFree(hostOutput);
  hipFree(resultRef);

  return 0;
}


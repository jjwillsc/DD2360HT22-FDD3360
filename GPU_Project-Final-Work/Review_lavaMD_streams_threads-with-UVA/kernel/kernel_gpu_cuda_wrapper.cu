//========================================================================================================================================================================================================200
//	DEFINE/INCLUDE
//========================================================================================================================================================================================================200

#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <stdlib.h>
#include <math.h>
 
#define DataType double

//======================================================================================================================================================150
//	MAIN FUNCTION HEADER
//======================================================================================================================================================150

#include "./../main.h"								// (in the main program folder)	needed to recognized input parameters


//======================================================================================================================================================150
//	UTILITIES
//======================================================================================================================================================150

#include "./../util/device/device.h"				// (in library path specified to compiler)	needed by for device functions
#include "./../util/timer/timer.h"					// (in library path specified to compiler)	needed by timer

//======================================================================================================================================================150
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION HEADER
//======================================================================================================================================================150

#include "./kernel_gpu_cuda_wrapper.h"				// (in the current directory)

//======================================================================================================================================================150
//	KERNEL
//======================================================================================================================================================150

#include "./kernel_gpu_cuda.cu"						// (in the current directory)	GPU kernel, cannot include with header file because of complications with passing of constant memory variables

//========================================================================================================================================================================================================200
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION
//========================================================================================================================================================================================================200

void 
kernel_gpu_cuda_wrapper(par_str par_cpu,
						dim_str dim_cpu,
						box_str* box_cpu,
						FOUR_VECTOR* rv_cpu,
						fp* qv_cpu,
						FOUR_VECTOR* fv_cpu)


{

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	// timer
	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;

	time0 = get_time();

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	hipDeviceSynchronize();

	//====================================================================================================100
	//	VARIABLES
	//====================================================================================================100

	box_str* d_box_gpu;
	FOUR_VECTOR* d_rv_gpu;
	fp* d_qv_gpu;
	FOUR_VECTOR* d_fv_gpu;

        int streamCount = 4;
        printf("CUDA Stream Count: %d\n", streamCount);

	dim3 threads;
	dim3 blocks;  

        //====================================================================================================100
        //      EXECUTION PARAMETERS
        //====================================================================================================100

        blocks.x = dim_cpu.number_boxes / (streamCount * NUMBER_THREADS);
        blocks.y = 1;
        threads.x = NUMBER_THREADS;                                                                                    // define the number of threads in the block
        threads.y = 1;

	time1 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY				(MALLOC)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

#if defined(USE_STREAM)

	hipMalloc((void**)&d_box_gpu, dim_cpu.box_mem);
	hipMalloc((void**)&d_rv_gpu, dim_cpu.space_mem);
	hipMalloc((void**)&d_qv_gpu, dim_cpu.space_mem2);
	hipMalloc((void**)&d_fv_gpu, dim_cpu.space_mem);

	hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * streamCount);

	for (int i = 0; i < streamCount; i++) {

		hipStreamCreate(&streams[i]);

                time2 = get_time();                 
	
                //hipMemcpyAsync(d_box_gpu, box_cpu, dim_cpu.box_mem, hipMemcpyHostToDevice, streams[i]);
                //hipMemcpyAsync(d_rv_gpu, rv_cpu, dim_cpu.space_mem, hipMemcpyHostToDevice, streams[i]);
                //hipMemcpyAsync(d_qv_gpu, qv_cpu, dim_cpu.space_mem2, hipMemcpyHostToDevice, streams[i]);
                //hipMemcpyAsync(d_fv_gpu, fv_cpu, dim_cpu.space_mem, hipMemcpyHostToDevice, streams[i]);

                hipMemcpyAsync(d_box_gpu, box_cpu, dim_cpu.box_mem, hipMemcpyDefault, streams[i]);
                hipMemcpyAsync(d_rv_gpu, rv_cpu, dim_cpu.space_mem, hipMemcpyDefault, streams[i]);
                hipMemcpyAsync(d_qv_gpu, qv_cpu, dim_cpu.space_mem2, hipMemcpyDefault, streams[i]);
                hipMemcpyAsync(d_fv_gpu, fv_cpu, dim_cpu.space_mem, hipMemcpyDefault, streams[i]);

                hipStreamSynchronize(streams[i]);

        }

        //======================================================================================================================================================150
        //      KERNEL
        //======================================================================================================================================================150

       	for (int i = 0; i < streamCount; i++) {

                time3 = get_time();

		// launch kernel - all boxes

                kernel_gpu_cuda<<<blocks, threads, 0, streams[i]>>>(par_cpu, dim_cpu, d_box_gpu, d_rv_gpu, d_qv_gpu, d_fv_gpu);

		checkCUDAError("Start");
        	hipDeviceSynchronize();

          	hipStreamSynchronize(streams[i]);

	}
 
        //======================================================================================================================================================150
        //      GPU MEMORY                      COPY (CONTD.)
        //======================================================================================================================================================150


        for (int i = 0; i < streamCount; i++) {

               	time4 = get_time();

                //hipMemcpyAsync(fv_cpu, d_fv_gpu, dim_cpu.space_mem, hipMemcpyDeviceToHost, streams[i]);

                hipMemcpyAsync(fv_cpu, d_fv_gpu, dim_cpu.space_mem, hipMemcpyDefault, streams[i]);

                hipStreamSynchronize(streams[i]);
        }


        //======================================================================================================================================================150
        //      GPU MEMORY DEALLOCATION
        //======================================================================================================================================================150


       	for (int i = 0; i < streamCount; i++) {
                hipStreamDestroy(streams[i]);
       	}
      	free(streams);  

#endif

	time5 = get_time();

	hipFree(d_rv_gpu);
	hipFree(d_qv_gpu);
	hipFree(d_fv_gpu);
	hipFree(d_box_gpu);

	time6 = get_time();	

	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150

	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) (time1-time0) / TIME_DIVIDER, (float) (time1-time0) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) (time2-time1) / TIME_DIVIDER, (float) (time2-time1) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) (time3-time2) / TIME_DIVIDER, (float) (time3-time2) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / TIME_DIVIDER, (float) (time4-time3) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) (time5-time4) / TIME_DIVIDER, (float) (time5-time4) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) (time6-time5) / TIME_DIVIDER, (float) (time6-time5) / (float) (time6-time0) * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) (time6-time0) / TIME_DIVIDER);

}
